#include "hip/hip_runtime.h"
#include "cuda/cu_matrix_functions.h"
#include <hip/hip_runtime.h>

#define CHECK_CUDA(func)                                                              \
{                                                                                     \
    hipError_t status = (func);                                                      \
    if (status != hipSuccess) {                                                      \
        printf("CUDA API failed at line %d with error %s (%d)\n",                     \
                __LINE__, hipGetErrorString(status), status);                        \
        return EXIT_FAILURE;                                                          \
    }                                                                                 \
}                                                                                     \

#define TILE_WIDTH 16                                                                 \

void checkCublasStatus(hipblasStatus_t status) {                                       
    if (status != HIPBLAS_STATUS_SUCCESS) {                                            
        std::cerr << "cuBLAS Error" << std::endl;                                     
        exit(EXIT_FAILURE);                                                   
    }                                                                                 
}   

__global__ void vectorAdd(const float*A, const float* B, float* C, int n) {
    int i = blockDim.x * blockIdx.x + threadIdx.x;

    if (i < n) {
        C[i] = A[i] + B[i];
    }
}

__global__ void simpleMMul2D(const float** A, const float** B, float** C, int blockWidth) {
    // using 2d coordinates for the matrix
    // TODO how do we specify a 2D matrix
    // We assume that the block is N x N
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    if ((row < blockWidth) && (col < blockWidth)) {
        float pValue = 0.f;
        for (int k = 0; k < blockWidth; ++k) {
            pValue += A[row][k] * B[k][col];
        }
        C[row][col] = pValue;
    }
}

__global__ void simpleMMul1D(const float* A, const float* B, float* C, int blockWidth) {
    // Because of the compute-to-global-memory ratio of this kernel the like occupancy 
    // will be around 2%
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if ((row < blockWidth) && (col < blockWidth)) {
        float pValue = 0.f;
        for (int k = 0; k < blockWidth; ++k) {
            pValue += A[row * blockWidth + k] * B[k * blockWidth + col];
        }
        C[row * blockWidth + col] = pValue;
    }
}

__global__ void tiledSquareMMul(const float* A, const float* B, float* C, int blockWidth) {
    // All threads in a block can acess 
    __shared__ float Mds[TILE_WIDTH][TILE_WIDTH];
    __shared__ float Nds[TILE_WIDTH][TILE_WIDTH];

    // need the x, y direction block location
    int bx = blockIdx.x; int by = blockIdx.y;
    // need the x, y direction thread locations in the block
    int tx = threadIdx.x; int ty = threadIdx.y;

    int row = by * blockWidth + ty;
    int col = bx * blockWidth + tx;

    // We first need to load the data into shared memory
    // count out the phases needed to compute all of the dot products
    float pValue = 0.f;
    for (int ph=0; ph < blockWidth / TILE_WIDTH; ++ph) {
        // load the data into shared memory
        // The portion of the matrix A needed will depend on the tileing phase
        // of the computation.
        Mds[ty][tx] = A[row * blockWidth + TILE_WIDTH * ph + tx];
        Nds[ty][tx] = B[(ph * TILE_WIDTH + ty) * blockWidth + col];
        // Doesn't affect the for loop within the kernel just the threads involved in 
        // the shared memory within the block.
        // We don't want the theads overwriting the content of shared memory
        // until we are finished using it.
        __syncthreads();
        // After the above syncthreads all of the data will be loaded into SM
        // Now compute the partial dot product of tx,ty
        for (int k = 0; k < TILE_WIDTH; ++k) {
            pValue += Mds[ty][k] * Nds[k][tx];
        }
        // make sure that all of the inner products have been computed before
        // starting to overwrite the data in shared memory
        __syncthreads();
    }
    C[row * blockWidth + col] = pValue;
}

__global__ void tiledSquareMMul2D(const float** A, const float** B, float** C, int blockWidth) {
    // setup a tile of shared memory
    __shared__ float Mds[TILE_WIDTH][TILE_WIDTH];
    __shared__ float Nds[TILE_WIDTH][TILE_WIDTH];

    // get the x,y direction of the block
    int bx = blockIdx.x; int by = blockIdx.y;
    // get the x, y direction of the thread within the block
    int tx = threadIdx.x; int ty = threadIdx.y;

    // compute the row and the column of the C matrix to be computed
    int row = by * blockWidth + ty;
    int col = bx * blockWidth + tx;

    float pValue = 0.f;
    // count out the phases necessary to compute the complete inner product.
    for (int ph = 0; ph < blockWidth / TILE_WIDTH; ++ph) {
        // use the threads to load the data into shared memory
        Mds[ty][tx] = A[row][ph * TILE_WIDTH + tx];
        Nds[ty][tx] = B[ph * TILE_WIDTH + ty][col];
        // synchronise the threads so that all of the data has been loaded into 
        // shared memory before computing the partial
        __syncthreads();
        // loop over all of the elements with the tile
        for (int k = 0; k < TILE_WIDTH; ++k) {
            pValue += Mds[ty][k] * Nds[k][tx];
        }
        // make sure that all of the inner products have been computed
        // before we start overwriting the data in shared memory
        __syncthreads();
    }
    C[row][col] = pValue;
}

namespace vector_kernels {

int vecAdd(float* A, float* B, float* C, int n) {
    /*
    Assuming that the host input memory has already been alocated.
    A is an input vector
    B is an input vector
    C is an output vector that holds the memory allocated to the solution
    */
    size_t size = n * sizeof(float);

    float* dA = NULL;
    CHECK_CUDA(hipMalloc((void**)&dA, n))
    
    float* dB = NULL;
    CHECK_CUDA(hipMalloc((void**)&dB, n))

    float* dC = NULL;
    CHECK_CUDA(hipMalloc((void**)&dC, n))

    // Copy the data from the host to the device vectors
    CHECK_CUDA(hipMemcpy(dA, A, size, hipMemcpyHostToDevice))
    CHECK_CUDA(hipMemcpy(dB, B, size, hipMemcpyHostToDevice))

    // perform the kernel operation

    int threadsPerBlock = 256;
    int blocksPerGrid = (n + threadsPerBlock - 1) / threadsPerBlock;
    printf("CUDA kernel launch with %d blocks of %d threads\n", blocksPerGrid, threadsPerBlock);
    vectorAdd<<<blocksPerGrid, threadsPerBlock>>>(dA, dB, dC, n);

    // copy the device memory for dC over to C
    CHECK_CUDA(hipMemcpy(C, dC, size, hipMemcpyDeviceToHost))

    // shut everything down and clean up 
    CHECK_CUDA(hipFree(dA))
    CHECK_CUDA(hipFree(dB))
    CHECK_CUDA(hipFree(dC))
    return 0;
}
}

namespace matrix_kernels 
{

/// @brief The host matrices are in row major format but they need to be converted into
/// column major format to use the cublas linear algebra functionality
/// takes the data in the Fmatrix and copies it over to the device
int FMatrix::toDevice() {
    CHECK_CUDA(hipMalloc((void**) dData, size));
    float h_ones[rows];
    for (int i = 0; i < rows; ++i) {
        h_ones[i] = 1.0f;
    }
    CHECK_CUDA(hipMemcpy(ones, h_ones, 
        rows * sizeof(float), hipMemcpyHostToDevice))
    //CHECK_CUDA(hipMemcpy(dData, data, size, hipMemcpyHostToDevice))
    checkCublasStatus(hipblasSetMatrix(rows, cols, sizeof(float), 
        data, cols, dData, rows));
}

/// @brief The device matrix is in column major format but it needs to be copied
/// back to row major format to work with numpy matrices. hipblasGetMatrix is used to
/// handle this operation
int FMatrix::copyToHost() {
    checkCublasStatus(hipblasGetMatrix(rows, cols, sizeof(float), 
        dData, rows, data, cols));
}

}

namespace implementation {

int printAttributes() {
    int devCount;
    CHECK_CUDA(hipGetDeviceCount(&devCount))
    for (int i=0; i < devCount; ++i) {
        hipDeviceProp_t prop;
        CHECK_CUDA(cudaGetDeviceProperties_v2(&prop, i));
        int max_threads = prop.maxThreadsPerBlock;
        int shared_mem = prop.sharedMemPerBlock;
        std::cout << "max threads per block: " << max_threads << std::endl;
        std::cout << "shared memory per block: " << shared_mem << " bytes" << std::endl;
    }
    return 0;
};
}